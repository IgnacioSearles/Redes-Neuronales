#include "hip/hip_runtime.h"
#include "redNeuronalGPU.h"

#include <iostream>
#include <random>
#include <fstream>

__global__ void realizarCalculosWeightsEnGPU(float* weights, float* nodos, int capa, int numWeights, int sizeAtCapa, int* prevNodosAtCapa, int* prevWeightsAtCapa) {
    int indice = blockIdx.x * blockDim.x + threadIdx.x;
    int paso = blockDim.x * gridDim.x;

    for (int weight = indice; weight < numWeights; weight += paso)
    {
        int nodoDeSalida = weight % sizeAtCapa;
        int nodoDeEntrada = weight / sizeAtCapa;
        float temp =  nodos[prevNodosAtCapa[capa - 1] + nodoDeEntrada] * weights[prevWeightsAtCapa[capa] + nodoDeEntrada * sizeAtCapa + nodoDeSalida];
        atomicAdd(&nodos[prevNodosAtCapa[capa] + nodoDeSalida], temp);                                                
    }
}

redNeuronal::redNeuronal(int mForma[], int mNumCapas, funcionesDeActivacion mActivacion, funcionesDeError mError)
{
    forma = mForma;
    numCapas = mNumCapas;
    funcionDeActivacion = mActivacion;
    funcionDeError = mError;

    initWeights();
    initBias();
    initNodos();

    objetivosActuales = (float*)malloc(forma[numCapas - 1] * sizeof(float));

    cambiarOffsetYMultiplicador(0.0001f, 1.0f);

    randomizarRed();
}

void redNeuronal::initWeights()
{
    numWeights = 0;

    for (int capa = 1; capa < numCapas; capa++)
    {
        numWeights += forma[capa - 1] * forma[capa];
    }
    hipMallocManaged(&weights, numWeights * sizeof(float));
    deltaWeights = (float *)malloc(numWeights * sizeof(float));

    hipMallocManaged(&prevWeightsAtCapa, numCapas * sizeof(int));
    prevWeightsAtCapa[0] = 0;
    prevWeightsAtCapa[1] = 0;
    for (int capa = 2; capa < numCapas; capa++)
    {
        prevWeightsAtCapa[capa] = forma[capa - 2] * forma[capa - 1] + prevWeightsAtCapa[capa - 1];
    }
}

void redNeuronal::initBias()
{
    numBias = 0;

    for (int capa = 1; capa < numCapas; capa++)
    {
        numBias += forma[capa];
    }
    biases = (float *)malloc(numBias * sizeof(float));
    deltaBiases = (float *)malloc(numBias * sizeof(float));

    prevBiasAtCapa = (int *)malloc(numCapas * sizeof(int));
    prevBiasAtCapa[0] = 0;
    prevBiasAtCapa[1] = 0;
    for (int capa = 2; capa < numCapas; capa++)
    {
        prevBiasAtCapa[capa] = forma[capa - 1] + prevBiasAtCapa[capa - 1];
    }
}

void redNeuronal::initNodos()
{
    numNodos = 0;

    for (int capa = 0; capa < numCapas; capa++)
    {
        numNodos += forma[capa];
    }
    hipMallocManaged(&nodos, numNodos * sizeof(float));

    hipMallocManaged(&prevNodosAtCapa, numCapas * sizeof(int));
    prevNodosAtCapa[0] = 0;
    for (int capa = 1; capa < numCapas; capa++)
    {
        prevNodosAtCapa[capa] = forma[capa - 1] + prevNodosAtCapa[capa - 1];
    }
}

void redNeuronal::randomizarRed()
{
    std::random_device generadorDeSemilla;
    std::mt19937 generador(generadorDeSemilla());
    std::uniform_real_distribution<float> distribucion(-1, 1);

    for (int weight = 0; weight < numWeights; weight++)
    {
        weights[weight] = distribucion(generador);
    }

    for (int bias = 0; bias < numBias; bias++)
    {
        biases[bias] = distribucion(generador);
    }
}

void redNeuronal::imprimirInfoPorConsola()
{
    std::cout << "Forma red: " << std::endl;
    for (int capa = 0; capa < numCapas; capa++)
        std::cout << "- " << forma[capa] << std::endl;
    std::cout << std::endl;

    std::cout << "Weights red: " << std::endl;
    for (int capa = 1; capa < numCapas; capa++)
    {
        for (int nodoDeSalida = 0; nodoDeSalida < forma[capa]; nodoDeSalida++)
        {
            for (int nodoDeEntrada = 0; nodoDeEntrada < forma[capa - 1]; nodoDeEntrada++)
            {
                std::cout << std::fixed << "Weight[" << capa << ", " << nodoDeSalida << ", " << nodoDeEntrada << "]: " << weights[prevWeightsAtCapa[capa] + nodoDeEntrada * forma[capa] + nodoDeSalida] << std::endl;
            }
        }
    }

    std::cout << "Biases red: " << std::endl;
    for (int capa = 1; capa < numCapas; capa++)
    {
        for (int nodoDeSalida = 0; nodoDeSalida < forma[capa]; nodoDeSalida++)
        {
            std::cout << "Bias[" << capa << ", " << nodoDeSalida << "]: " << biases[prevBiasAtCapa[capa] + nodoDeSalida] << std::endl;
        }
    }
}

void redNeuronal::cambiarOffsetYMultiplicador(float offset, float multiplicador) {
    OFFSET = offset;
    multiplicadorDeEntrenamiento = multiplicador;
}

float *redNeuronal::predecir(float *entradas)
{
    memcpy(nodos, entradas, forma[0] * sizeof(float));

    propagarEnGPU();

    float *salida = new float[forma[numCapas - 1]];
    memcpy(salida, nodos + prevNodosAtCapa[numCapas - 1], forma[numCapas - 1] * sizeof(float));
    return salida;
}

void redNeuronal::propagarEnGPU()
{
    for (int capa = 1; capa < numCapas; capa++)
    {
        for (int nodoDeSalida = 0; nodoDeSalida < forma[capa]; nodoDeSalida++)
        {
            nodos[prevNodosAtCapa[capa] + nodoDeSalida] = 0;
        }

        int bloqueSize = 256;
        int numBloques = (forma[capa] * forma[capa - 1] + bloqueSize - 1) / bloqueSize;

        realizarCalculosWeightsEnGPU<<<numBloques, bloqueSize>>>(weights, nodos, capa, forma[capa] * forma[capa - 1], forma[capa], prevNodosAtCapa, prevWeightsAtCapa);
        hipDeviceSynchronize();

        for (int nodoDeSalida = 0; nodoDeSalida < forma[capa]; nodoDeSalida++)
        {
            nodos[prevNodosAtCapa[capa] + nodoDeSalida] += biases[prevBiasAtCapa[capa] + nodoDeSalida];
            nodos[prevNodosAtCapa[capa] + nodoDeSalida] = aplicarFuncionDeActivacion(nodos[prevNodosAtCapa[capa] + nodoDeSalida]);
        }
    }
}

float redNeuronal::aplicarFuncionDeActivacion(const float& entrada) {
    switch (funcionDeActivacion)
    {
    case sigmoid:
        return 1.0f / (1.0f + exp(-entrada));
    }
    return -1.0f;
}

void redNeuronal::entrenar(float *ejemplos, float* objetivos, int numEjemplos) {
    //mezclarEjemplos(ejemplos, objetivos, numEjemplos);

    for (int ejemplo = 0; ejemplo < numEjemplos; ejemplo++) {
        memcpy(nodos, ejemplos + ejemplo * forma[0], forma[0] * sizeof(float));
        memcpy(objetivosActuales, objetivos + ejemplo * forma[numCapas - 1], forma[numCapas - 1] * sizeof(float));
                
        adjustarVariablesSegunEjemplo();

        std::cout << "\r" << ejemplo << "/" << numEjemplos;
    }
}

void redNeuronal::mezclarEjemplos(float* ejemplos, float* objetivos, int numEjemplos) {
    std::random_device generadorDeSemilla;
    std::mt19937 generador(generadorDeSemilla());
    std::uniform_int_distribution<int> distribucion(0, numEjemplos - 1);

    for (int ejemplo = 0; ejemplo < numEjemplos; ejemplo++) {
        int nuevaPos = distribucion(generador);
        float* tempEntradas = (float*) malloc(forma[0] * sizeof(float));
        memcpy(tempEntradas, ejemplos + nuevaPos * forma[0], forma[0] * sizeof(float));
        memcpy(ejemplos + nuevaPos * forma[0], ejemplos + ejemplo * forma[0], forma[0] * sizeof(float));
        memcpy(ejemplos + ejemplo * forma[0], tempEntradas, forma[0] * sizeof(float));

        float* tempSalidas = (float*) malloc(forma[numCapas - 1] * sizeof(float));
        memcpy(tempSalidas, objetivos + nuevaPos * forma[numCapas - 1], forma[numCapas - 1] * sizeof(float));
        memcpy(objetivos + nuevaPos * forma[numCapas - 1], objetivos + ejemplo * forma[numCapas - 1], forma[numCapas - 1] * sizeof(float));
        memcpy(objetivos + ejemplo * forma[numCapas - 1], tempSalidas, forma[numCapas - 1] * sizeof(float));
    }
}

void redNeuronal::adjustarVariablesSegunEjemplo() {
    errorPrediccionInicial = calcularError();

    for (int weight = 0; weight < numWeights; weight++) {
        calcularGradienteSegunError(weights[weight], deltaWeights[weight]);
    }

    for (int weight = 0; weight < numWeights; weight++) {
        weights[weight] += deltaWeights[weight];
    }

    errorPrediccionInicial = calcularError();

    for (int bias = 0; bias < numBias; bias++) {
        calcularGradienteSegunError(biases[bias], deltaBiases[bias]);
    }

    for (int bias = 0; bias < numBias; bias++) {
        biases[bias] += deltaBiases[bias];
    }
}

void redNeuronal::calcularGradienteSegunError(float& variable, float& deltaVariable) {
    variable += OFFSET;
    float errorPrediccionActual = calcularError();
    variable -= OFFSET;

    deltaVariable = ((errorPrediccionActual - errorPrediccionInicial) / OFFSET) * -multiplicadorDeEntrenamiento;
}

float redNeuronal::calcularError() {
    propagarEnGPU();

    float error = 0;
    for (int nodo = 0; nodo < forma[numCapas - 1]; nodo++) {
        error +=  aplicarFuncionDeError(objetivosActuales[nodo], nodos[prevNodosAtCapa[numCapas - 1] + nodo]);
    }

    error = aplicarFuncionDePromedioError(error);
    return error;
}

float redNeuronal::aplicarFuncionDeError(const float& actual, const float& prediccion) {
    switch (funcionDeError)
    {
    case meanSquared:
        return (actual - prediccion) * (actual - prediccion);
    case crossEntropy:
        return actual * log(prediccion) + (1 - actual) * log(1 - prediccion);
    }
    return -1.0f;
}

float redNeuronal::aplicarFuncionDePromedioError(const float& error) {
    switch (funcionDeError)
    {
    case meanSquared:
        return error * (1.0f / (2.0f * forma[numCapas - 1]));
    case crossEntropy:
        return error * -(1.0f / forma[numCapas - 1]);
    }
    return -1.0f;
}

void redNeuronal::guardarRedEnArchivo(const char* archivo) {
    std::ofstream archivoRedNeuronal;
    archivoRedNeuronal.open(archivo);
    archivoRedNeuronal.write((char*)weights, numWeights * sizeof(float));
    archivoRedNeuronal.write((char*)biases, numBias * sizeof(float));
    archivoRedNeuronal.close();
}

void redNeuronal::abrirRedDeArchivo(const char* archivo) {
    std::ifstream archivoRedNeuronal;
    archivoRedNeuronal.open(archivo);
    archivoRedNeuronal.read((char*)weights, numWeights * sizeof(float));
    archivoRedNeuronal.read((char*)biases, numBias * sizeof(float));
    archivoRedNeuronal.close();
}

redNeuronal::~redNeuronal()
{
    hipFree(prevWeightsAtCapa);
    free(prevBiasAtCapa);
    hipFree(prevNodosAtCapa);

    hipFree(weights);
    free(biases);
    hipFree(nodos);

    free(deltaWeights);
    free(deltaBiases);
    free(objetivosActuales);
}